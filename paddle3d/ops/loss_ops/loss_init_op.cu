#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/extension.h"

__global__ void init_idx( 
    const int n,
    const int len,
    int64_t* inputs) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int bid = blockIdx.y;
    if(bid < n && tid < len){
        int64_t* ptr = (int64_t*)inputs[bid];
        ptr[tid] = bid;
    }
}

std::vector<paddle::Tensor> loss_init_cuda(const paddle::Tensor& in){
    const int m = in.shape()[0];
    const int n = in.shape()[1];
   std::vector<paddle::Tensor> inputs(n);
   std::vector<int64_t*> input_ptrs(n);
   for(int i = 0; i < n; ++i){
    inputs[i] = paddle::empty({m, 1, 1}, paddle::DataType::INT64, paddle::GPUPlace());
    input_ptrs[i] = inputs[i].data<int64_t>();
   }
   paddle::Tensor d_ptrs = paddle::empty({n},
           paddle::DataType::INT64, paddle::GPUPlace());
   hipMemcpyAsync(d_ptrs.data<int64_t>(), input_ptrs.data(),
           sizeof(int64_t*) * n, hipMemcpyHostToDevice,
           inputs[0].stream()); 

   dim3 blocks(256, 1, 1);
   dim3 grids((m + 255) / 256, n, 1);  
   init_idx<<<grids, blocks, 0, inputs[0].stream()>>>(n, m,
           d_ptrs.data<int64_t>());
   return {paddle::experimental::concat(inputs, 1)}; 
}


